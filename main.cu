/* 
 * File:   main.cpp
 * Author: gregy
 *
 */

#include <cstdlib>
#include <vector>
#include <string>
#include <iostream>
#include <algorithm>
#include <fstream>
#include <string.h>
#include <omp.h>
#include <sstream>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <sys/time.h>
#define VTYPE int

using namespace std;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

class CLIArgumentsParser {
public:
    static char* getCmdOption(char ** begin, char ** end, const std::string & option)
	{
		char ** itr = std::find(begin, end, option);
		if (itr != end && ++itr != end)
		{
			return *itr;
		}
		return 0;
	}

	static bool cmdOptionExists(char** begin, char** end, const std::string& option)
	{
		return std::find(begin, end, option) != end;
	}
};

class CPU {
public:
	unsigned int startindex;
	unsigned int endindex;
	unsigned int tempindex;

	CPU(unsigned int startindex=0, unsigned int endindex=0, unsigned int tempindex = 0) {
		this->startindex = startindex;
		this->endindex = endindex;
		this->tempindex = tempindex;
	}
	void printData( int * data,char delim = ' ', ostream & out = cout) {
		for(unsigned int i=startindex; i< endindex; i++) {
			out <<  data[i] << delim;
		}
	}
	void printDataReverse(int * data, char delim = ' ', ostream & out = cout) {
		for(unsigned int i = endindex-1;i>=this->startindex && i<endindex; i--) {
			out << data[i] << delim;
		}
	}
};

struct Coordinates {
	int x;
	int y;
	int z;

	__host__ __device__ Coordinates(int x, int y, int z) {
		this->x = x;
		this->y = y;
		this->z = z;
	}
	__host__ __device__ Coordinates& operator+=(const Coordinates& rhs) {
		this->x += rhs.x;
		this->y += rhs.y;
		this->z += rhs.z;
		return *this;
	}
	 __host__ __device__ Coordinates& operator*=(const int& rhs) {
		this->x *= rhs;
		this->y *= rhs;
		this->z *= rhs;
		return *this;
	}
	__host__ __device__ Coordinates operator+(const Coordinates& rhs) const{
		Coordinates n = (*this);
		n += rhs;
	    return n;
	}
	__host__ __device__ Coordinates operator*(const int& rhs) const{
		Coordinates n = (*this);
		n *= rhs;
	    return n;
	}
};

static const Coordinates XVector(1,0,0);
static const Coordinates YVector(0,1,0);
static const Coordinates ZVector(0,0,1);

class CPUMatrix {
	size_t x,y,z;

public:
	size_t bucket;
	vector<VTYPE> *numbers;
    vector<CPU> CPUs;
	CPUMatrix(char * dimensions, vector<VTYPE> &numbers) {
		this->numbers = &numbers;
		stringstream ss;
		ss<< dimensions;
		int x = 1,y = 1,z = 1;
		ss >> x;
		if(ss.peek() == 'x') {
			ss.ignore();
		}
		else {
			throw "spatny format dimenzi mantice ma byt NxN[xN]";
		}
		ss >> y;
		if(ss.good()) {
			if(ss.peek() == 'x') {
				ss.ignore();
			}
			else {
				throw "spatny format dimenzi mantice ma byt NxN[xN]";
			}
			ss >> z;
		}
		cout << "Dimenze jsou " << x << "x" << y <<"x"<<z<<endl;
        if(x<1 || y<1 || z<1) {
            throw "Dimenze musi byt kladne!";
        }
        this->CPUs.resize(x*y*z);
		init(x,y,z, numbers);
	}
	CPUMatrix(int x, int y, int z, vector<VTYPE> &numbers): CPUs(x*y*z) {
       init(x,y,z,numbers); 
    }
	void init(int x, int y, int z, vector<VTYPE> &numbers) {
		this->x = x;
		this->y=y;
		this->z = z;
		
		if(numbers.size() < CPUs.size()) {
			cout << "Grid procesoru je vetsi nez pocet cisel...fail" << endl;
			throw "grid procesoru je vetsi nez pocet cisel";
		}
		size_t bucket = numbers.size()/this->CPUs.size()+ (numbers.size() % this->CPUs.size() != 0);
		this->bucket = bucket;

		int pos = 0;
		for(vector<CPU>::iterator i = this->CPUs.begin(); i!=this->CPUs.end(); ++i,++pos) {
			//handle crazy case where there are cpus with no numbers
			if(pos*bucket > numbers.size()) {
				cout << "Warning, cpus without work..." << endl;
				*i = *(new CPU(0, 0,pos*2*bucket));
			}
			else if((pos+1)*bucket > numbers.size()) {
				*i = *(new CPU(pos*bucket, numbers.size(), pos*2*bucket));
			}
			else {
				*i = *(new CPU(pos*bucket, (pos+1)*bucket, pos*2*bucket));
			}
		}
	}
    CPU* get(size_t x, size_t y, size_t z) {
		if(!this->coorValid(x,y,z)) {
			throw "mimo rozmer procesoru";
		}
        return &CPUs.at(x + y * this->x + z * this->x * this->y);
    }
    CPU* get(Coordinates c) {
        return this->get(c.x,c.y,c.z);
    }
	size_t getBucketSize() {
		return this->bucket;
	}
	bool coorValid(size_t x, size_t y, size_t z) {
		if(x>=this->x || y>=this->y || z>=this->z) {
			return false;
		}
		return true;
	}
	CPU* get(size_t i) {
		return (&CPUs.at(i));
	}
	size_t size(){
		return this->CPUs.size();
	}
	size_t sizeForVector(Coordinates vector) {
		return this->x*vector.x+this->y*vector.y+this->z*vector.z;
	}
	void printMatrix() {
		for(size_t z=0;z<this->z; z++) {
			cout << "-----------------------" << endl;
			for(int y = this->y-1;y>=0;y--) {
				cout << endl;
				for(size_t x=0;x<this->x;x++) {
					cout << "|";
					this->get(x,y,z)->printData(this->numbers->data());
					cout << "|";
				}
				cout << endl;
			}
			cout << "-----------------------" << endl;
		}
	}
};
size_t logbin(size_t number) {
	size_t ret = 0;
	while (number >>= 1) { ++ret; }
	return ret;
}

__device__ int getCpuIndex(Coordinates override = Coordinates(0,0,0)) {
	if(threadIdx.x+override.x >= blockDim.x || blockIdx.x+override.y >= gridDim.x || blockIdx.y+override.z >= gridDim.y) {
		return -1;
	}
	if(((int)threadIdx.x+override.x) < 0 || ((int)blockIdx.x+override.y) < 0 || ((int)blockIdx.y+override.z) < 0) {
		return -1;
	}
	return (threadIdx.x + override.x) + (blockIdx.x+override.y) * blockDim.x + (blockIdx.y+override.z) * gridDim.x * blockDim.x;
}
__device__ int getIndexForVector(Coordinates vector) {
	if(vector.x != 0)
		return threadIdx.x;
	if(vector.y != 0)
		return blockIdx.x;
	if(vector.z !=0)
		return blockIdx.y;
	
	return -1;
}

__device__ inline void mergeAndSplit(VTYPE* d_data, VTYPE* d_temp, size_t tempindex,size_t list1min, size_t list1max, size_t list2min, size_t list2max) {
	VTYPE * list3 = d_temp+tempindex;

	int index1 = list1min, index2 = list2min, index3 = 0;

    // Loop untill both arrays have reached their upper bound.
    while (index1 < list1max || index2 < list2max) {

        // Make sure the first array hasn't reached 
        // its upper bound already and make sure we 
        // don't compare outside bounds of the second 
        // array.
        // order of conditions very important, otherwise outside of bounds access!
        if (index2 >= list2max || (index1 < list1max && d_data[index1] <= d_data[index2]) ) {
            list3[index3] = d_data[index1];
            index1++;
        }
        else {
            list3[index3] = d_data[index2];
            index2++;
        }
        index3++;
    }
	index1 = list1min;
	index2 = list2min;
	index3 = 0;
	while(index1<list1max) {
		d_data[index1] = list3[index3];
		index1++;
		index3++;
	}
	while(index2<list2max) {
		d_data[index2] = list3[index3];
		index2++;
		index3++;
	}
}
__global__ void eotSortIterationCuda(CPU * d_cpu, VTYPE* d_data, Coordinates direction, VTYPE*d_temp,bool even, Coordinates snake = Coordinates(0,0,0), bool reverse = false, Coordinates skipvector = Coordinates(0,0,0)) {
	int myindex = getCpuIndex();
	if(snake.x == 1 || snake.y ==1|| snake.z ==1) {
		if(getIndexForVector(snake)%2 ==1) {
			direction = direction * -1;
		}
	}
	if(skipvector.x == 1 || skipvector.y ==1|| skipvector.z ==1) {
		if(getIndexForVector(skipvector)%2 ==0) {
			return;
		}
	}
    if(reverse) {
        direction = direction * -1;
    }
	int oindex = getCpuIndex(direction);
	if(oindex < 0) {
		return;
	}
	if(getIndexForVector(direction)%2 == 0 && even) {
		mergeAndSplit(d_data, d_temp,d_cpu[myindex].tempindex,d_cpu[myindex].startindex, d_cpu[myindex].endindex, d_cpu[oindex].startindex, d_cpu[oindex].endindex);
	}
	if(getIndexForVector(direction)%2 == 1 && !even) {
		mergeAndSplit(d_data, d_temp,d_cpu[myindex].tempindex,d_cpu[myindex].startindex, d_cpu[myindex].endindex, d_cpu[oindex].startindex, d_cpu[oindex].endindex);
	}
}

//knihovni implementace heapsortu
__global__ void localSort(CPU * d_cpu, VTYPE * d_data) {
	int vSize = d_cpu[getCpuIndex()].endindex - d_cpu[getCpuIndex()].startindex;
	VTYPE * vec = d_data+d_cpu[getCpuIndex()].startindex;
	int i;
    int originalVal;
    int promoteIndx;
    int parentIndx;

    for (i = 1; i < vSize; i++) {
        originalVal = vec[i];
        promoteIndx = i;
        parentIndx = (promoteIndx-1) / 2;
        while (promoteIndx > 0 && vec[parentIndx] < originalVal) {
            vec[promoteIndx] = vec[parentIndx];
            promoteIndx = parentIndx;
            parentIndx = (promoteIndx-1) / 2;
        }
        vec[promoteIndx] = originalVal;
    }

    int bottom;
    int displacedVal;
    int vacantNodeIndx;
    int leftIndx;
    int rightIndx;
    int maxIndx;

    for (bottom = vSize-1; bottom > 0; bottom--) {
        displacedVal = vec[bottom];
        vec[bottom] = vec[0];
        // ASSERT: Value in root moved to current bottom of vec

        vacantNodeIndx = 0;
        while (true) {
            leftIndx = 2*vacantNodeIndx + 1;
            if (leftIndx >= bottom)
                break;
            rightIndx = 2*vacantNodeIndx + 2;
            if (rightIndx >= bottom || vec[leftIndx] > vec[rightIndx])
                maxIndx = leftIndx;
            else
                maxIndx = rightIndx;
            if (vec[maxIndx] <= displacedVal)
                break;
            vec[vacantNodeIndx] = vec[maxIndx];
            vacantNodeIndx = maxIndx;
       }
       vec[vacantNodeIndx] = displacedVal;
       // ASSERT: Heap has been recreated in vec[0..bottom-1]
    }
}

void readFile(char * filename, vector<VTYPE> &data) {
	ifstream inputFile(filename, std::ifstream::in);

	if (inputFile) {        
    	VTYPE value;

		while ( inputFile >> value ) {
			data.push_back(value);
		}
        if(!inputFile.eof()) {
			throw "chyba nacitani dat ze souboru - fakt tam jsou jen cisla?";
        }
		inputFile.close();
    }
    else {
        throw "chyba nacitani dat ze souboru";
    }
}

void writeFile(char * filename, vector<VTYPE>& data) {
	streambuf * buf;
	ofstream of;

	if(filename) {
		of.open(filename);
		buf = of.rdbuf();
	} else {
		buf = std::cout.rdbuf();
	}

	ostream out(buf);
	
	if (out) {        
		for(vector<VTYPE>::iterator i = data.begin(); i!=data.end(); i++) {
			out << *i << endl;
		}
    }
    else {
        throw "chyba vypisu dat";
    }
}
void writeFileSnake(ostream &out, Coordinates plane, Coordinates rowvector, Coordinates colvector, CPUMatrix * matrix, bool reverse = false) {
	
	if (out) {
		size_t rows = matrix->sizeForVector(colvector);
		size_t cols = matrix->sizeForVector(rowvector);
		for(size_t row =0;row<rows;row++) {
			if(row%2==0) {
				for(size_t col = 0; col<cols;col++) {
                    if(reverse) {
						matrix->get(plane+rowvector*col+colvector*row)->printDataReverse(matrix->numbers->data(),'\n', out);
                    }
                    else {
						matrix->get(plane+rowvector*col+colvector*row)->printData(matrix->numbers->data(),'\n', out);
                    }
				}
			}
			else {
				for(size_t col = cols-1; col>0;col--) {
                    if(reverse)
						matrix->get(plane+rowvector*col+colvector*row)->printDataReverse(matrix->numbers->data(),'\n', out);
					else
						matrix->get(plane+rowvector*col+colvector*row)->printData(matrix->numbers->data(),'\n', out);
				}
                if(reverse)
					matrix->get(plane+colvector*row)->printDataReverse(matrix->numbers->data(),'\n', out);
                else
					matrix->get(plane+colvector*row)->printData(matrix->numbers->data(),'\n', out);
			}
		}

    }
    else {
        throw "chyba vypisu dat";
    }
}
void eotSortCuda(Coordinates dynvector, CPUMatrix &matrix, CPU * d_cpu, VTYPE * d_data, VTYPE * d_temp, Coordinates snake = Coordinates(0,0,0), bool reverse = false, Coordinates skipvector = Coordinates(0,0,0), size_t override_runs = 0) {

	size_t dimension_size = matrix.sizeForVector(dynvector);
    size_t runtimes = dimension_size/2+1;
	if(override_runs > 0) {
		runtimes = override_runs;
	}
	//TODO: prozkoumat to +1
	for(size_t count=0;count<runtimes;count++) {
		//even pairs
		eotSortIterationCuda<<<dim3(matrix.sizeForVector(YVector), matrix.sizeForVector(ZVector),1), matrix.sizeForVector(XVector)>>> (d_cpu,d_data, dynvector, d_temp,true, snake, reverse, skipvector);
		//odd pairs
		eotSortIterationCuda<<<dim3(matrix.sizeForVector(YVector), matrix.sizeForVector(ZVector),1), matrix.sizeForVector(XVector)>>> (d_cpu,d_data, dynvector, d_temp,false, snake, reverse, skipvector);
	}
}

void shearSortCuda(vector<VTYPE> &data, CPUMatrix &matrix,CPU*d_cpu,VTYPE* d_data,VTYPE* d_temp, Coordinates xvector = XVector, Coordinates yvector = YVector, bool reverse = false, Coordinates skipvector = Coordinates(0,0,0)) {
	size_t rowsize = matrix.sizeForVector(xvector);
	size_t colsize = matrix.sizeForVector(yvector);
	for(size_t count=0;count<logbin(colsize)+2;count++) {
		eotSortCuda(yvector,matrix,d_cpu,d_data, d_temp, Coordinates(0,0,0), reverse, skipvector);
		eotSortCuda(xvector,matrix,d_cpu,d_data, d_temp, yvector, reverse, skipvector);
	}

}
void dSortCuda(vector<VTYPE> &data, CPUMatrix &matrix,CPU*d_cpu,VTYPE* d_data,VTYPE* d_temp) {

	size_t xsize = matrix.sizeForVector(XVector);
	size_t ysize = matrix.sizeForVector(YVector);
	size_t zsize = matrix.sizeForVector(ZVector);
    Coordinates zero(0,0,0);
	shearSortCuda(data, matrix, d_cpu, d_data, d_temp, XVector, YVector);
	shearSortCuda(data, matrix, d_cpu, d_data, d_temp, ZVector, YVector);
	shearSortCuda(data, matrix, d_cpu, d_data, d_temp, XVector, ZVector, true);
	shearSortCuda(data, matrix, d_cpu, d_data, d_temp, XVector, ZVector, false, YVector);
	eotSortCuda(YVector,matrix,d_cpu,d_data, d_temp, Coordinates(0,0,0), false, Coordinates(0,0,0), 1);
	shearSortCuda(data, matrix, d_cpu, d_data, d_temp, XVector, ZVector, false);
}

int main(int argc, char** argv) {
	try {
		char * filename = CLIArgumentsParser::getCmdOption(argv, argv + argc, "-f");
		if (!filename) {
			cout << "Chybi argument -f se souborem cisel" << endl;
			return -1;
		}
		char * output = CLIArgumentsParser::getCmdOption(argv, argv + argc, "-o");
		streambuf * buf;
		ofstream of;

		if(output) {
			of.open(output);
			buf = of.rdbuf();
		} else {
			buf = std::cout.rdbuf();
		}

		ostream outputStream(buf);



		char * dimensions = CLIArgumentsParser::getCmdOption(argv, argv + argc, "-S");
		if(dimensions) {
			//SHEAR
			vector<VTYPE> data;
			readFile(filename, data);

			CPUMatrix matrix(dimensions, data);
            if(matrix.sizeForVector(ZVector) != 1) {
                cout << "Shear funguje jen na 2d mrizce..." << endl;
                return -1;
            }
            struct timeval time;
            gettimeofday(&time, NULL);
            double t1=time.tv_sec+(time.tv_usec/1000000.0);
			//CUDA INIT
			CPU * d_cpu;
			const size_t sz = size_t(matrix.CPUs.size()) * sizeof(CPU);
			gpuErrchk(hipMalloc((void**)&d_cpu, sz));
			hipMemcpy(d_cpu, matrix.CPUs.data(), sz, hipMemcpyHostToDevice);
			//copy data to gpu
			VTYPE * d_data;
			const size_t sd = size_t(data.size()) * sizeof(VTYPE);
			hipMalloc((void**)&d_data, sd);
			hipMemcpy(d_data, data.data(), sd, hipMemcpyHostToDevice);
			//docasny prostor pro merge
			VTYPE * d_temp;
			const size_t st = 2*matrix.bucket * sizeof(VTYPE)*matrix.CPUs.size();
			hipMalloc((void**)&d_temp, st);
			localSort<<<dim3(matrix.sizeForVector(YVector), matrix.sizeForVector(ZVector),1), matrix.sizeForVector(XVector)>>> (d_cpu,d_data);
			shearSortCuda(data, matrix,d_cpu,d_data,d_temp, XVector, YVector);

			hipMemcpy(data.data(), d_data, sd, hipMemcpyDeviceToHost);
            gettimeofday(&time, NULL);
            double t2=time.tv_sec+(time.tv_usec/1000000.0);
            printf("Sorting took %.6lf seconds\n", t2-t1);
			writeFileSnake(outputStream, Coordinates(0,0,0), XVector,YVector, &matrix);

			return 0;
		}
        
		
		dimensions = CLIArgumentsParser::getCmdOption(argv, argv + argc, "-3");
		if(dimensions) {
			//3DSort
			vector<VTYPE> data;
			readFile(filename, data);

			CPUMatrix matrix(dimensions, data);
            if(matrix.sizeForVector(ZVector) < 2) {
                cout << "3d sort potrebuje z souradnici" << endl;
                return -1;
            }
            struct timeval time;
            gettimeofday(&time, NULL);
            double t1=time.tv_sec+(time.tv_usec/1000000.0);
			//CUDA INIT
			CPU * d_cpu;
			const size_t sz = size_t(matrix.CPUs.size()) * sizeof(CPU);
			gpuErrchk(hipMalloc((void**)&d_cpu, sz));
			hipMemcpy(d_cpu, matrix.CPUs.data(), sz, hipMemcpyHostToDevice);
			//copy data to gpu
			VTYPE * d_data;
			const size_t sd = size_t(data.size()) * sizeof(VTYPE);
			hipMalloc((void**)&d_data, sd);
			hipMemcpy(d_data, data.data(), sd, hipMemcpyHostToDevice);
			//docasny prostor pro merge
			VTYPE * d_temp;
			const size_t st = 2*matrix.bucket * sizeof(VTYPE)*matrix.CPUs.size();
			hipMalloc((void**)&d_temp, st);
			localSort<<<dim3(matrix.sizeForVector(YVector), matrix.sizeForVector(ZVector),1), matrix.sizeForVector(XVector)>>> (d_cpu,d_data);
			dSortCuda(data, matrix,d_cpu,d_data,d_temp);

			hipMemcpy(data.data(), d_data, sd, hipMemcpyDeviceToHost);
            gettimeofday(&time, NULL);
            double t2=time.tv_sec+(time.tv_usec/1000000.0);
            printf("Sorting took %.6lf seconds\n", t2-t1);
            for(int y=0;y<matrix.sizeForVector(YVector);y++) {
				writeFileSnake(outputStream, Coordinates(0,y,0), XVector,ZVector, &matrix);
            }
			return 0;
		}

		cout << "Nespecifikovan ani shear (-S) ani 3dsort (-3). Syntax [-S|-3] NxN[xN] -- rozmer matice procesoru" << endl;
		return -1;

	}
	catch (char const * e) {
		cout << e << endl;
		return -1;
	}
} 